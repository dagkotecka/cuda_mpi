#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

int SIZE = 16;
int LIFE_CYCLES = 10;

#define DEBUG 0

typedef unsigned int uint;

enum error_case
{
	e_malloc,
	e_memcpyHtD,
	e_memcpyDtH,
	e_kernel,
	e_dealloc,
	e_reset,
	e_default = 99
};

#define XY(X, Y) (((X) * SIZE) + (Y))

#define valid(X, Y) (((X) < 0 || (X) >= SIZE || (Y) < 0 || (Y) >= SIZE) ? 0 : 1)

void check_error(hipError_t error, error_case place)
{
	if (error != hipSuccess)
	{
		switch (place)
		{
		case e_malloc:
			fprintf(stderr, "ERROR! CUDA malloc failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_memcpyHtD:
			fprintf(stderr, "ERROR! CUDA memcpy to device failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_memcpyDtH:
			fprintf(stderr, "ERROR! CUDA memcpy to host failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_kernel:
			fprintf(stderr, "ERROR! CUDA kernel function failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_dealloc:
			fprintf(stderr, "ERROR! CUDA deallocation failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_reset:
			fprintf(stderr, "ERROR! CUDA failed when tried to reset (error code %s)\n", hipGetErrorString(error));
			break;
		default:
			fprintf(stderr, "ERROR! CUDA unrecognized error (error code %s)\n", hipGetErrorString(error));
			break;
		}
		exit(EXIT_FAILURE);
	}
}

__global__ void
calculate_new_status(const unsigned int *board, unsigned int *new_board, const int SIZE)
{
	int aa = blockDim.x * blockIdx.x + threadIdx.x;

	if (aa < SIZE*SIZE)
	{
		int ii = aa / SIZE;
		int jj = aa % SIZE;
		int alive_neighbours = 0;

		alive_neighbours += (valid(ii - 1, jj - 1) ? board[XY(ii - 1, jj - 1)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii - 1, jj) ? board[XY(ii - 1, jj)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii - 1, jj + 1) ? board[XY(ii - 1, jj + 1)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii, jj - 1) ? board[XY(ii, jj - 1)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii, jj + 1) ? board[XY(ii, jj + 1)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii + 1, jj - 1) ? board[XY(ii + 1, jj - 1)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii + 1, jj) ? board[XY(ii + 1, jj)] : 0) ? 1 : 0;
		alive_neighbours += (valid(ii + 1, jj + 1) ? board[XY(ii + 1, jj + 1)] : 0) ? 1 : 0;

#if DEBUG
		new_board[XY(ii, jj)] = alive_neighbours;
#else
		unsigned int tmp_cell_val = 0;

		if (board[XY(ii, jj)] == 0)
		{
			if (alive_neighbours == 3)
			{
				tmp_cell_val = 1;
			}
			else
			{
				tmp_cell_val = 0;
			}
		}
		else
		{
			if (alive_neighbours < 2)
			{
				tmp_cell_val = 0;
			}
			else if (alive_neighbours > 3)
			{
				tmp_cell_val = 0;
			}
			else
			{
				tmp_cell_val = 1;
			}
		}

		new_board[XY(ii, jj)] = tmp_cell_val;
#endif // DEBUG
	}
}

extern "C" int* gameOfLife(unsigned int* cells, int size)
{
	if (argc < 3) exit(EXIT_FAILURE);
	else SIZE = size;

	int THREADS_PER_BLOCK = 16;
	int BLOCKS_PER_GRID = (SIZE*SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	hipError_t error = hipSuccess;
	srand(time(NULL));

	cells = (unsigned int*)malloc(SIZE * SIZE * sizeof(unsigned int*));

	if (cells == NULL)
	{
		fprintf(stderr, "Malloc failed!\n");
		exit(EXIT_FAILURE);
	}

	for (uint ii = 0; ii < SIZE; ii++)
	{
		for (uint jj = 0; jj < SIZE; jj++)
		{
			cells[XY(ii, jj)] = (unsigned int)rand() % 2;
		}
	}

	unsigned int *d_board = NULL;
	unsigned int *d_new_board = NULL;

	error = hipMalloc((void **)&d_board, SIZE*SIZE*sizeof(unsigned int));
	check_error(error, e_malloc);

	error = hipMalloc((void **)&d_new_board, SIZE*SIZE*sizeof(unsigned int));
	check_error(error, e_malloc);

	error = hipMemcpy(d_board, cells, SIZE*SIZE*sizeof(unsigned int), hipMemcpyHostToDevice);
	check_error(error, e_memcpyHtD);

	calculate_new_status << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(d_board, d_new_board, SIZE);
	error = hipGetLastError();
	check_error(error, e_kernel);

	error = hipMemcpy(cells, d_new_board, SIZE*SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);
	check_error(error, e_memcpyDtH);

	error = hipFree(d_board);
	check_error(error, e_dealloc);

	error = hipFree(d_new_board);
	check_error(error, e_dealloc);

	error = hipDeviceReset();
	check_error(error, e_reset);

	return cells;
}