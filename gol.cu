#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

__device__ int COLUMN_LEN = 0;
__device__ int ROW_LEN = 0;

#define DEBUG 0

typedef unsigned int uint;

enum error_case
{
	e_malloc,
	e_memcpyHtD,
	e_memcpyDtH,
	e_kernel,
	e_dealloc,
	e_reset,
	e_default = 99
};

#define XY(X, Y) (((X) * COLUMN_LEN) + (Y))

#define valid(X, Y) (((X) < 0 || (X) >= COLUMN_LEN || (Y) < 0 || (Y) >= ROW_LEN) ? 0 : 1)

void check_error(hipError_t error, error_case place)
{
	if (error != hipSuccess)
	{
		switch (place)
		{
		case e_malloc:
			fprintf(stderr, "ERROR! CUDA malloc failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_memcpyHtD:
			fprintf(stderr, "ERROR! CUDA memcpy to device failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_memcpyDtH:
			fprintf(stderr, "ERROR! CUDA memcpy to host failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_kernel:
			fprintf(stderr, "ERROR! CUDA kernel function failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_dealloc:
			fprintf(stderr, "ERROR! CUDA deallocation failed (error code %s)\n", hipGetErrorString(error));
			break;
		case e_reset:
			fprintf(stderr, "ERROR! CUDA failed when tried to reset (error code %s)\n", hipGetErrorString(error));
			break;
		default:
			fprintf(stderr, "ERROR! CUDA unrecognized error (error code %s)\n", hipGetErrorString(error));
			break;
		}
		exit(EXIT_FAILURE);
	}
}

__global__ void
calculate_new_status(const unsigned int *board, unsigned int *new_board, const int columnLen, const int rowLen)
{
	int aa = blockDim.x * blockIdx.x + threadIdx.x;

	if (aa < columnLen*rowLen)
	{
		int ii = aa / columnLen;
		int jj = aa % rowLen;
		int alive_neighbours = 0;

		if ((ii != 0) || (ii != (rowLen - 1)))
		{
			alive_neighbours += (valid(ii - 1, jj - 1) ? board[XY(ii - 1, jj - 1)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii - 1, jj) ? board[XY(ii - 1, jj)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii - 1, jj + 1) ? board[XY(ii - 1, jj + 1)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii, jj - 1) ? board[XY(ii, jj - 1)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii, jj + 1) ? board[XY(ii, jj + 1)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii + 1, jj - 1) ? board[XY(ii + 1, jj - 1)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii + 1, jj) ? board[XY(ii + 1, jj)] : 0) ? 1 : 0;
			alive_neighbours += (valid(ii + 1, jj + 1) ? board[XY(ii + 1, jj + 1)] : 0) ? 1 : 0;

#if DEBUG
			new_board[XY(ii, jj)] = alive_neighbours;
#else
			unsigned int tmp_cell_val = 0;

			if (board[XY(ii, jj)] == 0)
			{
				if (alive_neighbours == 3)
				{
					tmp_cell_val = 1;
				}
				else
				{
					tmp_cell_val = 0;
				}
			}
			else
			{
				if (alive_neighbours < 2)
				{
					tmp_cell_val = 0;
				}
				else if (alive_neighbours > 3)
				{
					tmp_cell_val = 0;
				}
				else
				{
					tmp_cell_val = 1;
				}
			}

			new_board[XY(ii, jj)] = tmp_cell_val;
#endif // DEBUG
		}
	}
}

extern "C" unsigned int* cudaCalculate(unsigned int * cells, unsigned int columnLen, unsigned int rowLen)
{
	if (columnLen < 3 || rowLen < 3) exit(EXIT_FAILURE);

	hipMemcpy(&COLUMN_LEN, &columnLen, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(&ROW_LEN, &rowLen, sizeof(unsigned int), hipMemcpyHostToDevice);

	int THREADS_PER_BLOCK = 16;
	int BLOCKS_PER_GRID = (columnLen*rowLen + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	hipError_t error = hipSuccess;
	srand(time(NULL));
	unsigned int *board = NULL;
	board = (unsigned int*)malloc(columnLen * rowLen * sizeof(unsigned int*));

	if (board == NULL)
	{
		fprintf(stderr, "Malloc failed!\n");
		exit(EXIT_FAILURE);
	}

	board = cells;

	unsigned int *d_board = NULL;
	unsigned int *d_new_board = NULL;

	error = hipMalloc((void **)&d_board, columnLen*rowLen*sizeof(unsigned int));
	check_error(error, e_malloc);

	error = hipMalloc((void **)&d_new_board, columnLen*rowLen*sizeof(unsigned int));
	check_error(error, e_malloc);

	error = hipMemcpy(d_board, board, columnLen*rowLen*sizeof(unsigned int), hipMemcpyHostToDevice);
	check_error(error, e_memcpyHtD);

	calculate_new_status << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> >(d_board, d_new_board, columnLen, rowLen);
	error = hipGetLastError();
	check_error(error, e_kernel);

	error = hipMemcpy(board, d_new_board, columnLen*rowLen*sizeof(unsigned int), hipMemcpyDeviceToHost);
	check_error(error, e_memcpyDtH);

	error = hipFree(d_board);
	check_error(error, e_dealloc);

	error = hipFree(d_new_board);
	check_error(error, e_dealloc);

	error = hipDeviceReset();
	check_error(error, e_reset);

	return board;
}